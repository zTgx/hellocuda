
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello from GPU\n");
}

int main(void) {

    printf("Hello from CPU\n");

    helloFromGPU<<< 1, 10 >>>();

    hipError_t e = hipDeviceReset();
    if (e != hipSuccess) {
        return -1;
    }

    return 0;
}